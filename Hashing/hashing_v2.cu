#include "hip/hip_runtime.h"
/*
    CUDA IMPLEMENTATION OF HASHTABLE:
    --> Kernels for simultaneous Insertion and Deletion of data are defined.
    --> For handling colisions, Open Addressing is used (Linear Probing).
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "Includes/gputimer.h"


#define TABLE_SIZE 1000
#define TABLE_BYTES TABLE_SIZE*sizeof(int)

#define INPUT_ARRAY_SIZE 100
#define INPUT_ARRAY_BYTES INPUT_ARRAY_SIZE*sizeof(int)
#define INPUT_THREDS_NUM 100
#define INPUT_BLOCKS_NUM (INPUT_ARRAY_SIZE/INPUT_THREDS_NUM)

#define DELETE_ARRAY_SIZE 10
#define DELETE_ARRAY_BYTES DELETE_ARRAY_SIZE*sizeof(int)
#define DELETE_THREDS_NUM 100
#define DELETE_BLOCKS_NUM (DELETE_ARRAY_SIZE/DELETE_THREDS_NUM)

#define NULL_DATA 0
#define DELETED_DATA INT_MAX

__device__ __host__ int hash(int n)
{
    /*
    A simple hash function to be used in our HashTable
    */
    return (n * n) % TABLE_SIZE;
}

__device__ __host__ void initializeHashTable(int *hm)
{
    for (int i = 0; i < TABLE_SIZE; i++)
    {
        hm[i] = NULL_DATA;
    }
}

void printHashTable(int *hm)
{
    /*
    This function will output the contents of a HashTable in the form:   KEY   -->   VALUE
    */
    printf("Printing the content of HashTable:-\n");
    printf("\tKEY\t-->\tVALUE\n");
    for (int i = 0; i < TABLE_SIZE; i++)
    {
        printf("\t%li\t-->\t", i);
        if(hm[i] == DELETED_DATA){
            printf("<Deleted>\n");
            continue;
        }
        if (hm[i] != NULL_DATA)
            printf("%lu\n", hm[i]);
        else
            printf("-\n");
    }
}

__global__ void parallel_insert(int *d_input, int *d_hm)
{
    /*
    To insert an array of data to HashTable all at once using GPU
    */
    int input_idx = blockIdx.x * INPUT_THREDS_NUM + threadIdx.x;
    int table_idx = hash(d_input[input_idx]);
    __syncthreads();
    while (table_idx < TABLE_SIZE){
        if(d_hm[table_idx] == NULL_DATA || d_hm[table_idx] == DELETED_DATA){
            atomicAdd(&d_hm[table_idx], d_input[input_idx]);
            return;
        }
    }
    if(table_idx == TABLE_SIZE){
        printf("No more Space! (Data = %li)\n", d_input[input_idx]);
        return;
    }
}

__global__ void parallel_delete(int *d_del, int* d_hm){
    int del_idx = blockIdx.x*DELETE_THREDS_NUM + threadIdx.x;
    int hm_idx = hash(d_del[del_idx]);
    while(hm_idx < TABLE_SIZE && d_hm[hm_idx] != NULL_DATA){
        if(d_hm[hm_idx] == d_del[del_idx]){
            d_hm[hm_idx] = DELETED_DATA;
            return;
        }
    }
    printf("Not Found!\n");
}

int main(int argc, char **argv)
{
    GpuTimer timer;

    // Variables in CPU
    printf("Initialising Hash-Table....\n");
    int *h_input = (int *)malloc(INPUT_ARRAY_BYTES); // Input Array
    int *h_delete = (int *)malloc(DELETE_ARRAY_BYTES); // Input Array
    int *h_hm = (int *)malloc(TABLE_BYTES); // Hashtable in Host Memory
    initializeHashTable(h_hm);

    // Variables in GPU
    int *d_input;
    int *d_delete;
    int *d_hm; // Hash Table in Device/GPU Memory
    hipMalloc((void **)&d_input, INPUT_ARRAY_BYTES);
    hipMalloc((void **)&d_delete, DELETE_ARRAY_BYTES);
    hipMalloc((void **)&d_hm, TABLE_BYTES);

    printf("\n");
    printf("Inserting an array sorted in ASCENDING ORDER...\n");
    for(int i = 0; i < INPUT_ARRAY_SIZE; i++){
        h_input[i] = i+1;
    }
    hipMemcpy(d_input, h_input, INPUT_ARRAY_BYTES, hipMemcpyHostToDevice);
    hipMemcpy(d_hm, h_hm, INPUT_ARRAY_BYTES, hipMemcpyHostToDevice);
    timer.Start();
    parallel_insert<<<INPUT_BLOCKS_NUM, INPUT_THREDS_NUM>>>(d_input, d_hm);
    timer.Stop();
    hipMemcpy(h_hm, d_hm, TABLE_BYTES, hipMemcpyDeviceToHost);
    printf("Insertion Completed!\n");
    printf("Time taken = %g ms\n", timer.Elapsed());

    printf("\n");
    // Printing the resulting HashTable
    printf("The Resulting Hash Table is saved in 'output.txt' file.\n");
    freopen("output.txt", "w", stdout);
    printHashTable(h_hm);

    free(h_input);
    free(h_delete);
    free(h_hm);
    hipFree(d_input);
    hipFree(d_delete);
    hipFree(d_hm);
    return 0;
}